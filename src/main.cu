#include <iostream>
#include "Engine.h"


int main(int argc, char** argv) {
	Engine engine;
	engine.init(1280, 720);
	engine.setBackgroundColor(0.2f, 0.3f, 0.3f, 1.f);
	engine.run();
	return 0;
}