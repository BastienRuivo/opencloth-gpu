#include "hip/hip_runtime.h"
#include "SolverExplicitGPU.h"
#include <hip/hip_runtime.h>
#include <iostream>
#define GLM_COMPILER 0
// ====================================
// kernel declaration
// ====================================
__global__
void solveGPU(glm::vec3 *position,
                glm::vec3 *velocity,
                glm::vec3 *acceleration,
                glm::vec3 *force,
                float *mass,
                glm::vec3 gravity,
                glm::vec3 wind,
                float viscosity,
                float deltaT,
                int size) {

    int gtid = blockIdx.x*blockDim.x+threadIdx.x;
    if (gtid < size) {
        if(mass[gtid] == 0.0f){
            acceleration[gtid] = glm::vec3(0.0f);
        }else{
            acceleration[gtid] = (force[gtid] / mass[gtid]) + gravity + wind;
        }
        velocity[gtid] = velocity[gtid] + deltaT * (acceleration[gtid] - viscosity * velocity[gtid]);
        position[gtid] = position[gtid] + deltaT * velocity[gtid];
        force[gtid] = glm::vec3(0.0f);
    }
}


__global__ void updateTableInt(int * table, int size) {
    int gtid = threadIdx.x ;
    if (gtid < size) {
        table[gtid] = gtid;
    }
}

SolverExplicitGPU::SolverExplicitGPU(
        const glm::vec3 & gravity, 
        const glm::vec3 &wind,
        float viscosity,
        float deltaT): Solver(gravity, wind, viscosity, deltaT) {
            
    
}


void SolverExplicitGPU::setData(
        std::vector<Spring*> * spring,
        std::vector<glm::vec3> * position, 
        std::vector<glm::vec3> * velocity, 
        std::vector<glm::vec3> * acceleration,
        std::vector<glm::vec3> * force, 
        std::vector<float> * mass) {
    
    this->springs = spring;
    this->position = position;
    this->velocity = velocity;
    this->acceleration = acceleration;
    this->force = force;
    this->mass = mass;
    
    int length = position->size();

    hipMalloc( (void**) &this->position_gpu, length * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->velocity_gpu, length * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->acceleration_gpu, length * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->force_gpu, length * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->mass_gpu, length * sizeof(float));
    hipMalloc( (void**) &this->springs_gpu, spring->size() * sizeof(Spring) );


    hipMemcpy(mass_gpu, &(*mass)[0], length * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(velocity_gpu, &(*velocity)[0], length * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(position_gpu, &(*position)[0], length * sizeof(glm::vec3), hipMemcpyHostToDevice);
}


void SolverExplicitGPU::solve(int tps) {
    int length = position->size();
    hipMemcpy(force_gpu, &(*force)[0], length * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(acceleration_gpu, &(*acceleration)[0], length * sizeof(glm::vec3), hipMemcpyHostToDevice);
    
    int blockSize = 1024;
    int gridSize = (int)ceil((float)length/blockSize);

    solveGPU<<<gridSize, blockSize>>>(position_gpu, velocity_gpu, 
                            acceleration_gpu, force_gpu, mass_gpu, gravity, wind, viscosity, deltaT, this->position->size());


    hipMemcpy(&(*force)[0], force_gpu, length * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(&(*position)[0], position_gpu, length * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(&(*acceleration)[0], acceleration_gpu, length * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    
}

void SolverExplicitGPU::updateSprings() {
    for(int i = 0; i < springs->size(); i++){
        int A = springs->at(i)->getParticleA();
        int B = springs->at(i)->getParticleB();

        glm::vec3 dPos = position->at(A) - position->at(B);
        glm::vec3 dVit = velocity->at(A) - velocity->at(B);
        glm::vec3 dPosNorm = glm::normalize(dPos);

        float diffLength = glm::length(dPos) - springs->at(i)->getParam()->GetRestLength();

        glm::vec3 fRaideur = springs->at(i)->getParam()->GetStiffness() * diffLength * dPosNorm;
        glm::vec3 fAmortissement = springs->at(i)->getParam()->GetDamping() * dPosNorm * glm::dot(dVit, dPosNorm);
        force->at(A) = force->at(A) - fRaideur - fAmortissement;
        force->at(B) = force->at(B) + fRaideur  + fAmortissement;
    }
}


void SolverExplicitGPU::update(int Tps){
    updateSprings();
    solve(Tps);
}