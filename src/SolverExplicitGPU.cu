#include "hip/hip_runtime.h"
#include "SolverExplicitGPU.h"
#include <hip/hip_runtime.h>
#include <iostream>
// ====================================
// kernel declaration
// ====================================
__global__ void solveGPU(float * vertex,
                Particle *particle,
                float3 *velocity,
                float3 *acceleration,
                float3 *force,
                float3 *partialForce,
                float *mass,
                float3 gravity,
                float3 wind,
                float viscosity,
                float deltaT,
                int size) {

    int gtid = blockIdx.x*blockDim.x+threadIdx.x;
    if (gtid < size) {
        for(int i = 0; i < particle[gtid].nbSpring; i++) {
            force[gtid] = force[gtid] + partialForce[particle[gtid].springs[i]] * particle[gtid].multiplier[i];
        }
        if(mass[gtid] == 0.0f){
            acceleration[gtid] = make_float3(0.f, 0.f, 0.f);
        }else{
            acceleration[gtid] = (force[gtid] / mass[gtid]) + gravity + wind;
        }
        velocity[gtid] = velocity[gtid] + deltaT * (acceleration[gtid] - viscosity * velocity[gtid]);
        vertex[gtid * 8] = vertex[gtid * 8] + deltaT * velocity[gtid].x;
        vertex[gtid * 8 + 1] = vertex[gtid * 8 + 1] + deltaT * velocity[gtid].y;
        vertex[gtid * 8 + 2] = vertex[gtid * 8 + 2] + deltaT * velocity[gtid].z;
        force[gtid] = make_float3(0.f, 0.f, 0.f);
    }
}

__global__ void updateSpringsGPU(float * vertex, float3 * velocity, Spring * springs, float3 * partialForce, int size) {  
    int gtid = blockIdx.x*blockDim.x+threadIdx.x;

    if(gtid < size){
        int A = springs[gtid].PA;
        int B = springs[gtid].PB;

        float3 dPos;
        dPos.x = vertex[A * 8] - vertex[B * 8];
        dPos.y = vertex[A * 8 + 1] - vertex[B * 8 + 1];
        dPos.z = vertex[A * 8 + 2] - vertex[B * 8 + 2];
        float3 dVit = velocity[A] - velocity[B];
        float3 dPosNorm = normalize(dPos);

        float diffLength = length(dPos) - springs[gtid].restLength;
        partialForce[gtid] = (springs[gtid].stiffness * diffLength * dPosNorm) + (springs[gtid].damping * dPos * dot(dVit, dPos));
    }
}


// ====================================
// CPU functions
// ====================================

SolverExplicitGPUData::SolverExplicitGPUData(glm::vec3 gravity, glm::vec3 wind, float viscosity, float deltaT,
        uint VBO,
        std::vector<glm::vec3> * velocity,
        std::vector<glm::vec3> * acceleration,
        std::vector<glm::vec3> * force,
        std::vector<Particle> * particles,
        std::vector<Spring> * spring,
        std::vector<float> * mass) : SolverData(gravity, wind, viscosity, deltaT) {
    this->particleCount = particles->size();
    this->springCount = spring->size();

    this->gravity_gpu = make_float3(gravity.x, gravity.y, gravity.z);
    this->wind_gpu = make_float3(wind.x, wind.y, wind.z);
    this->VBO = VBO;
    hipGraphicsGLRegisterBuffer(&this->cudaVboResource, this->VBO, cudaGraphicsMapFlagsNone);
    

    hipMalloc( (void**) &this->vertex, this->particleCount * sizeof(float) * 8 );
    hipMalloc( (void**) &this->velocity, this->particleCount * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->partialForce, this->particleCount * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->acceleration, this->particleCount * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->force, this->particleCount * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->mass, this->particleCount * sizeof(float));
    hipMalloc( (void**) &this->springs, this->springCount * sizeof(Spring) );
    hipMalloc( (void**) &this->partialForce, this->springCount * sizeof(glm::vec3));
    hipMalloc( (void**) &this->particles, this->particleCount * sizeof(Particle));


    hipMemcpy(this->mass,          &(*mass)[0],            this->particleCount * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->velocity,      &(*velocity)[0],        this->particleCount * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(this->force,         &(*force)[0],           this->particleCount * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(this->acceleration,  &(*acceleration)[0],    this->particleCount * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(this->springs,       &(*spring)[0],          this->springCount * sizeof(Spring), hipMemcpyHostToDevice);
    hipMemcpy(this->particles,     &(*particles)[0],       this->particleCount * sizeof(Particle), hipMemcpyHostToDevice);
    hipGraphicsMapResources(1, &cudaVboResource, 0);
}

void printCudaLog() {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}

SolverExplicitGPUData::~SolverExplicitGPUData() {
    hipGraphicsUnmapResources(1, &cudaVboResource, 0);
    hipGraphicsUnregisterResource(cudaVboResource);
    hipFree(velocity);
    hipFree(acceleration);
    hipFree(force);
    hipFree(mass);
    hipFree(springs);
    hipFree(particles);
    hipFree(partialForce);
}

SolverExplicitGPU::SolverExplicitGPU(SolverExplicitGPUData * data) : _data(data) {
    type = SolverType::SOLVER_GPU;
}

void SolverExplicitGPU::solve(int tps) {
    int blockSize = 1024;
    int gridSize = (int)ceil((float)_data->particleCount/blockSize);
    
    

    solveGPU<<<gridSize, blockSize>>>(_data->vertex, _data->particles, _data->velocity, 
        _data->acceleration, _data->force, _data->partialForce, 
        _data->mass, _data->gravity_gpu, _data->wind_gpu, 
        _data->viscosity, _data->deltaT, _data->particleCount);
}

void SolverExplicitGPU::updateSprings() {
    int blockSize = 1024;
    int gridSize = (int)ceil((float)_data->springCount/blockSize);
    updateSpringsGPU<<<gridSize, blockSize>>>(_data->vertex, _data->velocity, _data->springs, _data->partialForce, _data->springCount);
}



void SolverExplicitGPU::update(int Tps){
    hipStream_t stream;
    
    hipGraphicsResourceGetMappedPointer((void**)&(_data->vertex), &(_data->vertexSize), (_data->cudaVboResource));
    updateSprings();
    solve(Tps);
    //hipDeviceSynchronize();
    hipStreamSynchronize(stream);
    

}

SolverExplicitGPU::~SolverExplicitGPU() {
    delete _data;
}