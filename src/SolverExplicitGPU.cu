#include "hip/hip_runtime.h"
#include "SolverExplicitGPU.h"
#include <hip/hip_runtime.h>
#include <iostream>
#define GLM_COMPILER 0
// ====================================
// kernel declaration
// ====================================
__global__ void solveGPU(float * vertex,
                Particle *particle,
                glm::vec3 *velocity,
                glm::vec3 *acceleration,
                glm::vec3 *force,
                glm::vec3 *partialForce,
                float *mass,
                glm::vec3 gravity,
                glm::vec3 wind,
                float viscosity,
                float deltaT,
                int size) {

    int gtid = blockIdx.x*blockDim.x+threadIdx.x;
    if (gtid < size) {
        for(int i = 0; i < particle[gtid].nbSpring; i++) {
            if(particle[gtid].springs[i] != -1) {
                if(particle[gtid].isNegative[i])
                    force[gtid] = force[gtid] - partialForce[particle[gtid].springs[i]];
                else
                    force[gtid] = force[gtid] + partialForce[particle[gtid].springs[i]];
            }
        }
        if(mass[gtid] == 0.0f){
            acceleration[gtid] = glm::vec3(0.0f);
        }else{
            acceleration[gtid] = (force[gtid] / mass[gtid]) + gravity + wind;
        }
        velocity[gtid] = velocity[gtid] + deltaT * (acceleration[gtid] - viscosity * velocity[gtid]);

        vertex[gtid * 8] = vertex[gtid * 8] + deltaT * velocity[gtid].x;
        vertex[gtid * 8 + 1] = vertex[gtid * 8 + 1] + deltaT * velocity[gtid].y;
        vertex[gtid * 8 + 2] = vertex[gtid * 8 + 2] + deltaT * velocity[gtid].z;

        force[gtid] = glm::vec3(0.0f);
    }
}

__global__ void updateSpringsGPU(float * vertex, glm::vec3 * velocity, Spring * springs, glm::vec3 * partialForce, int size) {  
    int gtid = blockIdx.x*blockDim.x+threadIdx.x;

    if(gtid < size){
        int A = springs[gtid].PA;
        int B = springs[gtid].PB;

        glm::vec3 dPos;
        dPos.x = vertex[A * 8] - vertex[B * 8];
        dPos.y = vertex[A * 8 + 1] - vertex[B * 8 + 1];
        dPos.z = vertex[A * 8 + 2] - vertex[B * 8 + 2];
        glm::vec3 dVit = velocity[A] - velocity[B];
        glm::vec3 dPosNorm = glm::normalize(dPos);

        float diffLength = glm::length(dPos) - springs[gtid].restLength;
        partialForce[gtid] = (springs[gtid].stiffness * diffLength * dPosNorm) + (springs[gtid].damping * dPos * glm::dot(dVit, dPos));
    }
}


// ====================================
// CPU functions
// ====================================

SolverExplicitGPUData::SolverExplicitGPUData(glm::vec3 gravity, glm::vec3 wind, float viscosity, float deltaT,
        std::vector<float> * vertex,
        std::vector<glm::vec3> * velocity,
        std::vector<glm::vec3> * acceleration,
        std::vector<glm::vec3> * force,
        std::vector<Particle> * particles,
        std::vector<Spring> * spring,
        std::vector<float> * mass) : SolverData(gravity, wind, viscosity, deltaT) {
    this->particleCount = particles->size();
    this->springCount = spring->size();

    this->vertex_cpu = vertex;

    hipMalloc( (void**) &this->vertex, this->particleCount * sizeof(float) * 8 );
    hipMalloc( (void**) &this->velocity, this->particleCount * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->partialForce, this->particleCount * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->acceleration, this->particleCount * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->force, this->particleCount * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->mass, this->particleCount * sizeof(float));
    hipMalloc( (void**) &this->springs, this->springCount * sizeof(Spring) );
    hipMalloc( (void**) &this->partialForce, this->springCount * sizeof(glm::vec3));
    hipMalloc( (void**) &this->particles, this->particleCount * sizeof(Particle));


    hipMemcpy(this->mass,          &(*mass)[0],            this->particleCount * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->velocity,      &(*velocity)[0],        this->particleCount * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(this->vertex,        &(*vertex)[0],          this->particleCount * sizeof(float) * 8, hipMemcpyHostToDevice);
    hipMemcpy(this->force,         &(*force)[0],           this->particleCount * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(this->acceleration,  &(*acceleration)[0],    this->particleCount * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(this->springs,       &(*spring)[0],          this->springCount * sizeof(Spring), hipMemcpyHostToDevice);
    hipMemcpy(this->particles,     &(*particles)[0],       this->particleCount * sizeof(Particle), hipMemcpyHostToDevice);
}

SolverExplicitGPUData::~SolverExplicitGPUData() {
    hipFree(vertex);
    hipFree(velocity);
    hipFree(acceleration);
    hipFree(force);
    hipFree(mass);
    hipFree(springs);
    hipFree(particles);
    hipFree(partialForce);
}

SolverExplicitGPU::SolverExplicitGPU(SolverExplicitGPUData * data) : _data(data) {}

void SolverExplicitGPU::solve(int tps) {
    int blockSize = 1024;
    int gridSize = (int)ceil((float)_data->particleCount/blockSize);

    solveGPU<<<gridSize, blockSize>>>(_data->vertex, _data->particles, _data->velocity, 
        _data->acceleration, _data->force, _data->partialForce, 
        _data->mass, _data->gravity, _data->wind, 
        _data->viscosity, _data->deltaT, _data->particleCount);

    hipMemcpy(&(*_data->vertex_cpu)[0], _data->vertex, _data->particleCount * 8 * sizeof(float), hipMemcpyDeviceToHost);
}

void SolverExplicitGPU::updateSprings() {
    int blockSize = 1024;
    int gridSize = (int)ceil((float)_data->springCount/blockSize);

    updateSpringsGPU<<<gridSize, blockSize>>>(_data->vertex, _data->velocity, _data->springs, _data->partialForce, _data->springCount);
}



void SolverExplicitGPU::update(int Tps){
    updateSprings();
    solve(Tps);
}

SolverExplicitGPU::~SolverExplicitGPU() {
    delete _data;
}