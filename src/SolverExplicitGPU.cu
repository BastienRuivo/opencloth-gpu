#include "hip/hip_runtime.h"
#include "SolverExplicitGPU.h"
#include <hip/hip_runtime.h>
#include <iostream>
#define GLM_COMPILER 0
// ====================================
// kernel declaration
// ====================================
__global__
void solveGPU(glm::vec3 *position,
                glm::vec3 *velocity,
                glm::vec3 *acceleration,
                glm::vec3 *force,
                float *mass,
                glm::vec3 gravity,
                glm::vec3 wind,
                float viscosity,
                float deltaT,
                int size) {

    int gtid = blockIdx.x*blockDim.x+threadIdx.x;
    if (gtid < size) {
        if(mass[gtid] == 0.0f){
            acceleration[gtid] = glm::vec3(0.0f);
        }else{
            acceleration[gtid] = (force[gtid] / mass[gtid]) + gravity + wind;
        }
        velocity[gtid] = velocity[gtid] + deltaT * (acceleration[gtid] - viscosity * velocity[gtid]);
        position[gtid] = position[gtid] + deltaT * velocity[gtid];
        force[gtid] = glm::vec3(0.0f);
    }
}

__global__ void updateSpringsGPU(glm::vec3 * position, glm::vec3 * velocity, Spring * springs, ForceToAdd * result, int size) {  
    int gtid = blockIdx.x*blockDim.x+threadIdx.x;

    if(gtid < size){
        int A = springs[gtid].PA;
        int B = springs[gtid].PB;

        glm::vec3 dPos = position[A] - position[B];
        glm::vec3 dVit = velocity[A] - velocity[B];
        glm::vec3 dPosNorm = glm::normalize(dPos);

        float diffLength = glm::length(dPos) - springs[gtid].restLength;

        glm::vec3 fRaideur = springs[gtid].stiffness * diffLength * dPosNorm;
        glm::vec3 fAmortissement = springs[gtid].damping * dPosNorm * glm::dot(dVit, dPosNorm);

        result[gtid] = {A, B, fRaideur - fAmortissement}; 
    }
}

SolverExplicitGPU::SolverExplicitGPU(
        const glm::vec3 & gravity, 
        const glm::vec3 &wind,
        float viscosity,
        float deltaT): Solver(gravity, wind, viscosity, deltaT) {
            
    
}


void SolverExplicitGPU::setData(
        std::vector<Spring> * spring,
        std::vector<glm::vec3> * position, 
        std::vector<glm::vec3> * velocity, 
        std::vector<glm::vec3> * acceleration,
        std::vector<glm::vec3> * force, 
        std::vector<float> * mass) {
    
    this->springs = spring;
    this->position = position;
    this->velocity = velocity;
    this->acceleration = acceleration;
    this->force = force;
    this->mass = mass;
    
    this->forcesToAdd.resize(springs->size());
    for (int i = 0; i < springs->size(); i++)
    {
        this->forcesToAdd[i] = {0, 0, glm::vec3(0.0f)};
    }
    

    int length = position->size();

    hipMalloc( (void**) &this->position_gpu, length * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->velocity_gpu, length * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->acceleration_gpu, length * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->force_gpu, length * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->mass_gpu, length * sizeof(float));
    hipMalloc( (void**) &this->springs_gpu, spring->size() * sizeof(Spring) );
    hipMalloc( (void**) &this->forcesToAdd_gpu, spring->size() * sizeof(ForceToAdd));


    hipMemcpy(mass_gpu, &(*mass)[0], length * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(velocity_gpu, &(*velocity)[0], length * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(position_gpu, &(*position)[0], length * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(forcesToAdd_gpu, &forcesToAdd[0], spring->size() * sizeof(ForceToAdd), hipMemcpyHostToDevice);
}


void SolverExplicitGPU::solve(int tps) {
    int length = position->size();
    hipMemcpy(force_gpu, &(*force)[0], length * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(acceleration_gpu, &(*acceleration)[0], length * sizeof(glm::vec3), hipMemcpyHostToDevice);
    
    int blockSize = 1024;
    int gridSize = (int)ceil((float)length/blockSize);

    solveGPU<<<gridSize, blockSize>>>(position_gpu, velocity_gpu, 
                            acceleration_gpu, force_gpu, mass_gpu, gravity, wind, viscosity, deltaT, this->position->size());


    hipMemcpy(&(*force)[0], force_gpu, length * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(&(*position)[0], position_gpu, length * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(&(*acceleration)[0], acceleration_gpu, length * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    
}

void SolverExplicitGPU::updateSprings() {
    int length = springs->size();
    hipMemcpy(springs_gpu, &(*springs)[0], length * sizeof(Spring), hipMemcpyHostToDevice);

    int blockSize = 1024;
    int gridSize = (int)ceil((float)length/blockSize);

    updateSpringsGPU<<<gridSize, blockSize>>>(position_gpu, velocity_gpu, springs_gpu, forcesToAdd_gpu, length);

    hipMemcpy(&forcesToAdd[0], forcesToAdd_gpu, length * sizeof(ForceToAdd), hipMemcpyDeviceToHost);

    for(int i = 0; i < length; i++){
        force->at(forcesToAdd[i].A) -= forcesToAdd[i].force;
        force->at(forcesToAdd[i].B) += forcesToAdd[i].force;
    }
}



void SolverExplicitGPU::update(int Tps){
    updateSprings();
    solve(Tps);
}