#include "hip/hip_runtime.h"
#include "SolverExplicitGPU.h"
#include <hip/hip_runtime.h>
#include <iostream>
#define GLM_COMPILER 0
// ====================================
// kernel declaration
// ====================================
__global__
void solveGPU(float * vertex,
                Particle *particle,
                glm::vec3 *velocity,
                glm::vec3 *acceleration,
                glm::vec3 *force,
                glm::vec3 *partialForce,
                float *mass,
                glm::vec3 gravity,
                glm::vec3 wind,
                float viscosity,
                float deltaT,
                int size) {

    int gtid = blockIdx.x*blockDim.x+threadIdx.x;
    if (gtid < size) {
        for(int i = 0; i < particle[gtid].nbSpring; i++) {
            if(particle[gtid].springs[i] != -1) {
                if(particle[gtid].isNegative[i])
                    force[gtid] = force[gtid] - partialForce[particle[gtid].springs[i]];
                else
                    force[gtid] = force[gtid] + partialForce[particle[gtid].springs[i]];
            }
        }
        if(mass[gtid] == 0.0f){
            acceleration[gtid] = glm::vec3(0.0f);
        }else{
            acceleration[gtid] = (force[gtid] / mass[gtid]) + gravity + wind;
        }
        velocity[gtid] = velocity[gtid] + deltaT * (acceleration[gtid] - viscosity * velocity[gtid]);

        vertex[gtid * 8] = vertex[gtid * 8] + deltaT * velocity[gtid].x;
        vertex[gtid * 8 + 1] = vertex[gtid * 8 + 1] + deltaT * velocity[gtid].y;
        vertex[gtid * 8 + 2] = vertex[gtid * 8 + 2] + deltaT * velocity[gtid].z;

        force[gtid] = glm::vec3(0.0f);
    }
}

__global__ void updateSpringsGPU(float * vertex, glm::vec3 * velocity, Spring * springs, glm::vec3 * partialForce, int size) {  
    int gtid = blockIdx.x*blockDim.x+threadIdx.x;

    if(gtid < size){
        int A = springs[gtid].PA;
        int B = springs[gtid].PB;

        glm::vec3 dPos;
        dPos.x = vertex[A * 8] - vertex[B * 8];
        dPos.y = vertex[A * 8 + 1] - vertex[B * 8 + 1];
        dPos.z = vertex[A * 8 + 2] - vertex[B * 8 + 2];
        glm::vec3 dVit = velocity[A] - velocity[B];
        glm::vec3 dPosNorm = glm::normalize(dPos);

        float diffLength = glm::length(dPos) - springs[gtid].restLength;
        partialForce[gtid] = (springs[gtid].stiffness * diffLength * dPosNorm) + (springs[gtid].damping * dPos * glm::dot(dVit, dPos));
    }
}

SolverExplicitGPU::SolverExplicitGPU(
        const glm::vec3 & gravity, 
        const glm::vec3 &wind,
        float viscosity,
        float deltaT): Solver(gravity, wind, viscosity, deltaT) {
            
    
}


void SolverExplicitGPU::setData(
        std::vector<Spring> * spring,
        std::vector<float> * vertex, 
        std::vector<Particle> * particles,
        std::vector<glm::vec3> * velocity, 
        std::vector<glm::vec3> * acceleration,
        std::vector<glm::vec3> * force, 
        std::vector<float> * mass) {
    
    this->springs = spring;
    this->vertex = vertex;
    this->particles = particles;
    this->velocity = velocity;
    this->acceleration = acceleration;
    this->force = force;
    this->mass = mass;

    
    
    this->partialForce = new std::vector<glm::vec3>(spring->size());


    int length = particles->size();

    hipMalloc( (void**) &this->vertex_gpu, length * sizeof(float) * 8 );
    hipMalloc( (void**) &this->velocity_gpu, length * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->partialForce_gpu, length * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->acceleration_gpu, length * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->force_gpu, length * sizeof(glm::vec3) );
    hipMalloc( (void**) &this->mass_gpu, length * sizeof(float));
    hipMalloc( (void**) &this->springs_gpu, spring->size() * sizeof(Spring) );
    hipMalloc( (void**) &this->partialForce_gpu, spring->size() * sizeof(glm::vec3));
    hipMalloc( (void**) &this->particles_gpu, length * sizeof(Particle));


    hipMemcpy(mass_gpu, &(*mass)[0], length * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(velocity_gpu, &(*velocity)[0], length * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(vertex_gpu, &(*vertex)[0], length * sizeof(float) * 8, hipMemcpyHostToDevice);
    hipMemcpy(force_gpu, &(*force)[0], length * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(acceleration_gpu, &(*acceleration)[0], length * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(springs_gpu, &(*springs)[0], spring->size() * sizeof(Spring), hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu, &(*particles)[0], length * sizeof(Particle), hipMemcpyHostToDevice);
}


void SolverExplicitGPU::solve(int tps) {
    int length = particles->size();
    
    
    int blockSize = 1024;
    int gridSize = (int)ceil((float)length/blockSize);

    solveGPU<<<gridSize, blockSize>>>(vertex_gpu, particles_gpu, velocity_gpu, 
                            acceleration_gpu, force_gpu, partialForce_gpu, mass_gpu, gravity, wind, viscosity, deltaT, length);

    //hipMemcpy(&(*vertex)[0], vertex_gpu, length * 8 * sizeof(float), hipMemcpyDeviceToHost);
}

void SolverExplicitGPU::updateSprings() {
    int length = springs->size();
    int blockSize = 1024;
    int gridSize = (int)ceil((float)length/blockSize);

    updateSpringsGPU<<<gridSize, blockSize>>>(vertex_gpu, velocity_gpu, springs_gpu, partialForce_gpu, length);
}



void SolverExplicitGPU::update(int Tps){
    updateSprings();
    solve(Tps);
}

SolverExplicitGPU::~SolverExplicitGPU() {
    hipFree(vertex_gpu);
    hipFree(velocity_gpu);
    hipFree(acceleration_gpu);
    hipFree(force_gpu);
    hipFree(mass_gpu);
    hipFree(springs_gpu);
    hipFree(particles_gpu);
    hipFree(partialForce_gpu);
}